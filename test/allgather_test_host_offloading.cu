#include "hip/hip_runtime.h"
#include <mscclpp/core.hpp>
#include <mscclpp/fifo.hpp>
#include <mscclpp/proxy.hpp>
#include <mscclpp/epoch.hpp>

#ifdef MSCCLPP_USE_MPI_FOR_TESTS
#include "mpi.h"
#endif // MSCCLPP_USE_MPI_FOR_TESTS
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <unistd.h>
#include <unordered_map>

int nranksPerNode;
int rank;
int world_size;

// Propagate errors up

#define MSCCLPPCHECK(call)                                                                                             \
  do {                                                                                                                 \
    mscclppResult_t res = call;                                                                                        \
    if (res != mscclppSuccess && res != mscclppInProgress) {                                                           \
      /* Print the back trace*/                                                                                        \
      printf("Failure at %s:%d -> %s\n", __FILE__, __LINE__, mscclppGetErrorString(res));                              \
      return res;                                                                                                      \
    }                                                                                                                  \
  } while (0)

// Check CUDA RT calls
#define CUDACHECK(cmd)                                                                                                 \
  do {                                                                                                                 \
    hipError_t err = cmd;                                                                                             \
    if (err != hipSuccess) {                                                                                          \
      printf("%s:%d Cuda failure '%s'\n", __FILE__, __LINE__, hipGetErrorString(err));                                \
      exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (false)

// Measure current time in second.
static double getTime(void)
{
  struct timespec tspec;
  if (clock_gettime(CLOCK_MONOTONIC, &tspec) == -1) {
    printf("clock_gettime failed\n");
    exit(EXIT_FAILURE);
  }
  return (tspec.tv_nsec / 1.0e9) + tspec.tv_sec;
}


__global__ void kernel(int r, int nranks, mscclpp::DeviceProxyFifo fifo, mscclpp::DeviceEpoch::DeviceHandle* handles)
{
  int tid = threadIdx.x;
  if (tid != r)
    handles[tid].epochIncrement();
  if (tid == 0){
    mscclpp::ProxyTrigger trigger;
    trigger.fst = 1;
    fifo.push(trigger);
  }
  if (tid != r)
    handles[tid].wait();
}

int rankToLocalRank(int rank)
{
  return rank % nranksPerNode;
}

int rankToNode(int rank)
{
  return rank / nranksPerNode;
}

void print_usage(const char* prog)
{
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  printf("usage: %s IP:PORT [rank nranks]\n", prog);
#else
  printf("usage: %s IP:PORT rank nranks\n", prog);
#endif
}

void initializeAndAllocateAllGatherData(int rank, int world_size, size_t dataSize, size_t nelemsPerGPU, int** data_h,
                                        int** data_d)
{
  CUDACHECK(hipMalloc(data_d, dataSize));
  CUDACHECK(hipMemset(*data_d, 0, dataSize));

  *data_h = new int[nelemsPerGPU * world_size];
  for (size_t i = 0; i < nelemsPerGPU * world_size; i++) {
    int val = i + 1;
    if (i / nelemsPerGPU == (size_t)rank) {
      (*data_h)[i] = val;
    } else {
      (*data_h)[i] = 0;
    }
  }
  CUDACHECK(hipMemcpy(*data_d, *data_h, dataSize, hipMemcpyHostToDevice));
}

class MyProxyService {
private:
  int deviceNumaNode;
public:
  MyProxyService() : remoteMemories(world_size), connections(world_size),
  proxy([&](mscclpp::ProxyTrigger triggerRaw) { return handleTrigger(triggerRaw); }, [&]() { bindThread(); }) {
    // int cudaDevice;
    // CUDACHECK(hipGetDevice(&cudaDevice));
    // getDeviceNumaNode(cudaDevice, &deviceNumaNode);
  }

  void bindThread() {
    // if (deviceNumaNode >= 0) {
    //   numaBind(deviceNumaNode);
    //   INFO(MSCCLPP_INIT, "NUMA node of DeviceChannelService proxy thread is set to %d", deviceNumaNode);
    // }
  }

  mscclpp::ProxyHandlerResult handleTrigger(mscclpp::ProxyTrigger triggerRaw) {
    if (triggerRaw.fst == 1) {
      int dataSizePerRank = dataSize / world_size;
      for (int r = 0; r < world_size; ++r) {
        if (r == rank) {
          continue;
        }
        connections[r]->write(remoteMemories[r], rank*dataSizePerRank, localMemory, rank*dataSizePerRank, dataSizePerRank);
        deviceEpochs[r]->signal();
        connections[r]->flush();
      }
    }
    return mscclpp::ProxyHandlerResult::FlushFifoTailAndContinue;
  }
  mscclpp::Proxy proxy;
  std::vector<mscclpp::RegisteredMemory> remoteMemories;
  mscclpp::RegisteredMemory localMemory;
  std::vector<std::shared_ptr<mscclpp::HostEpoch>> hostEpochs;
  std::vector<std::shared_ptr<mscclpp::DeviceEpoch>> deviceEpochs;
  std::vector<std::shared_ptr<mscclpp::Connection>> connections;
  int dataSize; 
};

void setupProxyService(mscclpp::Communicator& comm, MyProxyService& proxyService, int* data_d, int dataSize)
{
  proxyService.dataSize = dataSize;
  int thisNode = rankToNode(rank);
  int cudaNum = rankToLocalRank(rank);
  std::string ibDevStr = "mlx5_ib" + std::to_string(cudaNum);
  mscclpp::Transport ibTransport = mscclpp::getIBTransportByDeviceName(ibDevStr);
  std::vector<mscclpp::NonblockingFuture<mscclpp::RegisteredMemory>> remoteMemories(world_size);

  proxyService.localMemory = comm.registerMemory(data_d, dataSize, mscclpp::Transport::CudaIpc | ibTransport);
  for (int r = 0; r < world_size; ++r) {
    if (r == rank){
      proxyService.hostEpochs.emplace_back(nullptr);
      proxyService.deviceEpochs.emplace_back(nullptr);
      continue;
    }
    mscclpp::Transport transport;
    if (rankToNode(r) == thisNode) {
      transport = mscclpp::Transport::CudaIpc;
    } else {
      transport = ibTransport;
    }
    // Connect with all other ranks
    proxyService.connections[r] = comm.connectOnSetup(r, 0, transport);
    if (rankToNode(r) == thisNode) {
      proxyService.hostEpochs.emplace_back(nullptr);
    } else {
      proxyService.hostEpochs.emplace_back(std::make_shared<mscclpp::HostEpoch>(comm, proxyService.connections[r]));
    }
    proxyService.deviceEpochs.emplace_back(std::make_shared<mscclpp::DeviceEpoch>(comm, proxyService.connections[r]));
    comm.sendMemoryOnSetup(proxyService.localMemory, r, 0);

    remoteMemories[r] = comm.recvMemoryOnSetup(r, 0);
  }

  comm.setup();
  for (int r = 0; r < world_size; ++r) {
    if (r == rank){
      continue;
    }
    proxyService.remoteMemories[r] = remoteMemories[r].get();
  }
}

std::unordered_map<std::string, std::string> parseArgs(int argc, char* argv[])
{
  std::unordered_map<std::string, std::string> options;

  for (int i = 1; i < argc; i++) {
    std::string arg = argv[i];
    if (arg == "-datasize") {
      if (i + 1 < argc) {
        options["datasize"] = argv[++i];
      } else {
        fprintf(stderr, "Error: -datasize option requires an argument.\n");
        exit(-1);
      }
    } else if (arg == "-help" || arg == "-h") {
      exit(0);
    } else {
      fprintf(stderr, "Error: Unknown option %s\n", argv[i]);
      exit(-1);
    }
  }
  return options;
}


int main(int argc, char* argv[])
{
  // sleep(10);
  MPI_Init(&argc, &argv);
  auto parsedArgs = parseArgs(argc, argv);

  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  // get the local number of nodes with MPI
  MPI_Comm shmcomm;
  MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL, &shmcomm);
  int shmrank;
  MPI_Comm_size(shmcomm, &shmrank);
  nranksPerNode = shmrank;
  MPI_Comm_free(&shmcomm);


  int cudaNum = rankToLocalRank(rank);
  CUDACHECK(hipSetDevice(cudaNum));

  if (rank == 0)
    printf("Initializing MSCCL++\n");
  auto bootstrap = std::make_shared<mscclpp::Bootstrap>(rank, world_size);
  mscclpp::UniqueId uniqueId;
  if (rank == 0)
    uniqueId = bootstrap->createUniqueId();
  MPI_Bcast(&uniqueId, sizeof(uniqueId), MPI_BYTE, 0, MPI_COMM_WORLD);
  bootstrap->initialize(uniqueId);
  mscclpp::Communicator comm(bootstrap);

  int* data_d;
  int* data_h;
  size_t dataSize = 1024 * 1024 * 1024;
  if (parsedArgs.find("datasize") != parsedArgs.end()) {
    dataSize = std::stoul(parsedArgs["datasize"]);
  }
  size_t nelemsPerGPU = dataSize / sizeof(int) / world_size;

  if (rank == 0)
    printf("Initializing data for allgather test\n");
  initializeAndAllocateAllGatherData(rank, world_size, dataSize, nelemsPerGPU, &data_h, &data_d);

  if (rank == 0)
    printf("Setting up the connection in MSCCL++\n");

  MyProxyService proxyService;
  setupProxyService(comm, proxyService, data_d, dataSize);

  if (rank == 0)
    printf("Launching MSCCL++ proxy threads\n");
  proxyService.proxy.start();
  mscclpp::DeviceProxyFifo fifo = proxyService.proxy.fifo().deviceFifo();
  if (rank == 0)
    printf("Testing the correctness of AllGather implementation\n");
  hipStream_t stream;
  CUDACHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  mscclpp::DeviceEpoch::DeviceHandle* deviceHandles;

  CUDACHECK(hipMalloc(&deviceHandles, sizeof(mscclpp::DeviceEpoch::DeviceHandle) * world_size));
  for (int i = 0; i < world_size; ++i) {
    if (i == rank)
      continue;
    auto handle = proxyService.deviceEpochs[i]->deviceHandle();
    CUDACHECK(hipMemcpy(&deviceHandles[i], &handle, sizeof(mscclpp::DeviceEpoch::DeviceHandle), hipMemcpyHostToDevice));
  }

  kernel<<<1, world_size, 0, stream>>>(rank, world_size, fifo, deviceHandles);
  CUDACHECK(hipStreamSynchronize(stream));

  CUDACHECK(hipMemcpy(data_h, data_d, dataSize, hipMemcpyDeviceToHost));

  for (size_t i = 0; i < nelemsPerGPU * world_size; i++) {
    int val = i + 1;
    if (data_h[i] != val) {
      printf("oh uh! data_h[%ld] (%d) != val (%d)\n", i, data_h[i], val);
      break;
    }
  }

  bootstrap->barrier();
  if (rank == 0)
    printf("Correctness test passed!\n");

  double t0, t1, ms, time_in_us;
  int iterwithoutcudagraph = 10;
  if (rank == 0)
    printf("Running %d iterations of the kernel without CUDA graph\n", iterwithoutcudagraph);
  CUDACHECK(hipStreamSynchronize(stream));
  bootstrap->barrier();
  t0 = getTime();
  for (int i = 0; i < iterwithoutcudagraph; ++i) {
    kernel<<<1, world_size, 0, stream>>>(rank, world_size, fifo, deviceHandles);
  }
  CUDACHECK(hipStreamSynchronize(stream));
  bootstrap->barrier();
  t1 = getTime();
  ms = (t1 - t0) * 1000.0;
  time_in_us = ms * 1000. / (float)iterwithoutcudagraph;
  printf("No Graph %d report: size %lu time: %f us/iter algBW %f GBps\n", rank, dataSize, time_in_us,
          (double)(dataSize) / 1e9 / (time_in_us / 1e6));

  // cudaGraph Capture
  int cudagraphiter = 10;
  if (rank == 0)
    printf("Capturing %d iterations of the kernel in a CUDA graph\n", cudagraphiter);
  hipGraph_t graph;
  hipGraphExec_t instance;
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  for (int i = 0; i < cudagraphiter; ++i) {
    kernel<<<1, world_size, 0, stream>>>(rank, world_size, fifo, deviceHandles);
  }
  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

  int cudagraphwarmup = 10;
  if (rank == 0)
    printf("Warming up %d iterations of the CUDA graph with %d iterations of the kernel\n", cudagraphwarmup,
            cudagraphiter);
  for (int i = 0; i < cudagraphwarmup; ++i) {
    hipGraphLaunch(instance, stream);
  }
  CUDACHECK(hipStreamSynchronize(stream));

  // measure runtime
  int cudagraphlaunch = 10;
  if (rank == 0)
    printf("Running %d iterations of the CUDA graph with %d iterations of the kernel\n", cudagraphlaunch,
            cudagraphiter);
  bootstrap->barrier();
  t0 = getTime();
  for (int i = 0; i < cudagraphlaunch; ++i) {
    hipGraphLaunch(instance, stream);
  }
  CUDACHECK(hipStreamSynchronize(stream));

  t1 = getTime();
  ms = (t1 - t0) * 1000.0;
  time_in_us = ms * 1000. / (float)cudagraphlaunch / (float)cudagraphiter;
  printf("Rank %d report: size %lu time: %f us/iter algBW %f GBps\n", rank, dataSize, time_in_us,
          (double)(dataSize) / 1e9 / (time_in_us / 1e6));
  bootstrap->barrier();

  if (rank == 0)
    printf("Stopping MSCCL++ proxy threads\n");
  proxyService.proxy.stop();



#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  MPI_Finalize();
#endif
  return 0;
}
