#include "hip/hip_runtime.h"
#include "mscclpp.h"
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
#include "mpi.h"
#endif // MSCCLPP_USE_MPI_FOR_TESTS
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string>

#define RANKS_PER_NODE 8
#define TEST_CONN_TYPE 1 // 0: P2P(for local)+IB(for remote), 1: IB-Only

// Check CUDA RT calls
#define CUDACHECK(cmd) do {                                   \
    hipError_t err = cmd;                                    \
    if( err != hipSuccess ) {                                \
        printf("%s:%d Cuda failure '%s'", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while(false)

__global__ void kernel(mscclppDevConn_t devConns, int rank, int world_size)
{
  int warpId = threadIdx.x / 32;
  int remoteRank = (warpId < rank) ? warpId : warpId + 1;
  mscclppDevConn_t devConn = &devConns[(remoteRank < rank) ? remoteRank : remoteRank - 1];
  volatile int *data = (volatile int *)devConn->localBuff;
  volatile int *localFlag = devConn->localFlag;
  volatile int *remoteFlag = devConn->remoteFlag;
  volatile uint64_t *trig = (volatile uint64_t *)devConn->trigger;

  if (threadIdx.x == 0) {
    // Set my data and flag
    *(data + rank) = rank + 1;
    __threadfence_system();
    *localFlag = 1;
  }
  __syncthreads();

  // Each warp receives data from different ranks
  if (threadIdx.x % 32 == 0) {
    if (devConn->remoteBuff == NULL) { // IB
      // Trigger sending data and flag
      uint64_t dataOffset = rank * sizeof(int);
      uint64_t dataSize = sizeof(int);
      *trig = (dataOffset << 32) + dataSize;

      // Wait until the proxy have sent my data and flag
      while (*trig != 0) {}

      // Wait for receiving data from remote rank
      while (*remoteFlag != 1) {}
    } else { // P2P
      // Directly read data
      volatile int *remoteData = (volatile int *)devConn->remoteBuff;

      // Wait until the remote data is set
      while (*remoteFlag != 1) {}

      // Read remote data
      data[remoteRank] = remoteData[remoteRank];
    }
  }
}

int rankToLocalRank(int rank)
{
  return rank % RANKS_PER_NODE;
}

int rankToNode(int rank)
{
  return rank / RANKS_PER_NODE;
}

void print_usage(const char *prog)
{
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  printf("usage: %s IP:PORT [rank nranks]\n", prog);
#else
  printf("usage: %s IP:PORT rank nranks\n", prog);
#endif
}

int main(int argc, const char *argv[])
{
#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  if (argc != 2 && argc != 4) {
    print_usage(argv[0]);
    return -1;
  }
  const char *ip_port = argv[1];
  int rank;
  int world_size;
  if (argc == 4) {
    rank = atoi(argv[2]);
    world_size = atoi(argv[3]);
  } else {
    MPI_Init(NULL, NULL);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  }
#else
  if (argc != 4) {
    print_usage(argv[0]);
    return -1;
  }
  const char *ip_port = argv[1];
  int rank = atoi(argv[2]);
  int world_size = atoi(argv[3]);
#endif
  int localRank = rankToLocalRank(rank);
  int thisNode = rankToNode(rank);

  mscclppComm_t comm;
  mscclppResult_t res = mscclppCommInitRank(&comm, world_size, rank, ip_port);
  if (res != mscclppSuccess) {
    printf("mscclppCommInitRank failed\n");
    return -1;
  }

  CUDACHECK(hipSetDevice(localRank));

  int *data_d;
  int *flag_d;
  CUDACHECK(hipMalloc(&data_d, sizeof(int) * world_size));
  CUDACHECK(hipMalloc(&flag_d, sizeof(int)));
  CUDACHECK(hipMemset(data_d, 0, sizeof(int) * world_size));
  CUDACHECK(hipMemset(flag_d, 0, sizeof(int)));

  std::string ibDevStr = "mlx5_ib" + std::to_string(localRank);

  for (int r = 0; r < world_size; ++r) {
    if (r == rank) continue;
    mscclppTransport_t transportType = mscclppTransportIB;
    const char *ibDev = ibDevStr.c_str();
#if (TEST_CONN_TYPE == 0) // P2P+IB
    if (rankToNode(r) == thisNode) {
      transportType = mscclppTransportP2P;
      ibDev = NULL;
    }
#endif
    // Connect with all other ranks
    res = mscclppConnect(comm, r, data_d, sizeof(int) * world_size, flag_d, 0, transportType, ibDev);
    if (res != mscclppSuccess) {
      printf("mscclppConnect failed\n");
      return -1;
    }
  }

  res = mscclppConnectionSetup(comm);
  if (res != mscclppSuccess) {
    printf("mscclppConnectionSetup failed\n");
    return -1;
  }

  res = mscclppProxyLaunch(comm);
  if (res != mscclppSuccess) {
    printf("mscclppProxyLaunch failed\n");
    return -1;
  }

  mscclppDevConn_t devConns;
  mscclppGetDevConns(comm, &devConns);

  kernel<<<1, 32 * (world_size - 1)>>>(devConns, rank, world_size);
  CUDACHECK(hipDeviceSynchronize());

  res = mscclppProxyStop(comm);
  if (res != mscclppSuccess) {
    printf("mscclppProxyStop failed\n");
    return -1;
  }

  // Read results from GPU
  int *buf = (int *)calloc(world_size, sizeof(int));
  if (buf == nullptr) {
    printf("calloc failed\n");
    return -1;
  }
  CUDACHECK(hipMemcpy(buf, data_d, sizeof(int) * world_size, hipMemcpyDeviceToHost));

  bool failed = false;
  for (int i = 0; i < world_size; ++i) {
    if (buf[i] != i + 1) {
      printf("rank: %d, wrong data: %d, expected %d\n", rank, buf[i], i + 1);
      failed = true;
    }
  }
  if (failed) {
    return -1;
  }

  res = mscclppCommDestroy(comm);
  if (res != mscclppSuccess) {
    printf("mscclppDestroy failed\n");
    return -1;
  }

#ifdef MSCCLPP_USE_MPI_FOR_TESTS
  if (argc == 2) {
    MPI_Finalize();
  }
#endif
  printf("Succeeded! %d\n", rank);
  return 0;
}
